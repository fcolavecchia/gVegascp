#include "hip/hip_runtime.h"
#include "vegasconst.h"
#define HIP_PI_F 3.141592654f

__device__
float sum(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += rx[i];
	}
	value = 1.f / sqrt((float)dim/12.f) * (value - (float)dim / 2.f);
	return value;
}

__device__
float sqsum(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += rx[i] * rx[i];
	}
	value = sqrtf(45.f / (4.f * (float)dim)) * (value - (float)dim / 3);
	return value;
}

__device__
float sumsqroot(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += sqrtf(rx[i]);
	}
	value = sqrtf(18.f / (float)dim) * (value - 2.f/3.f * (float)dim);
	return value;
}

__device__
float prodones(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= copysignf(1.f, rx[i]-0.5f);
	}
	return value;
}

__device__
float prodexp(float* rx, int dim)
{
	float e = sqrtf((15.f * expf(15.f) + 15.f) / (13.f * expf(15.f) + 17.f));
	e = powf(e, float(dim) * 0.5f);
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= ((expf(30.f * rx[i] - 15.f)) - 1.f) / (expf(30.f * rx[i] - 15.f) + 1.f);		
	}
	value *= e;
	return value;
}

