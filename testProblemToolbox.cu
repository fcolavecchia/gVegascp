#include "hip/hip_runtime.h"
#include "vegasconst.h"
#define HIP_PI_F 3.141592654f

__device__
float sum(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += rx[i];
	}
	value = 1.f / sqrt((float)dim/12.f) * (value - (float)dim / 2.f);
	return value;
}

__device__
float sqsum(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += rx[i] * rx[i];
	}
	value = sqrtf(45.f / (4.f * (float)dim)) * (value - (float)dim / 3);
	return value;
}

__device__
float sumsqroot(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += sqrtf(rx[i]);
	}
	value = sqrtf(18.f / (float)dim) * (value - 2.f/3.f * (float)dim);
	return value;
}

__device__
float prodones(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= copysignf(1.f, rx[i]-0.5f);
	}
	return value;
}

__device__
float prodexp(float* rx, int dim)
{
	float e = sqrtf((15.f * expf(15.f) + 15.f) / (13.f * expf(15.f) + 17.f));
	e = powf(e, float(dim) * 0.5f);
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= ((expf(30.f * rx[i] - 15.f)) - 1.f) / (expf(30.f * rx[i] - 15.f) + 1.f);		
	}
	value *= e;
	return value;
}

__device__
float prodcub(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= (-2.4f*sqrtf(7.f)*(rx[0]-0.5f)+8.f*sqrtf(7.f)*(rx[i]-0.5f)*(rx[i]-0.5f)*(rx[i]-0.5f));
	}
	return value;
}

__device__
//PRODX has a lot of extremes when dimensions are big, it's expected to not do well
float prodx(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= (r[x] - 0.5f);
	}
	value *= powf(2.f*sqrtf(3.f), (float) dim);
	return value;
}

__device__
float sumfifj(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		float aux = 0.f;
		for (int j = 0; j < i; j++){
			aux += copysignf(1.f,(1.f/6.f - rx[j])*( rx[j] - 4.f/6.f));
		}
		value += copysignf(1.f,(1.f/6.f - rx[i])*( rx[i] - 4.f/6.f)) * aux;
	}
	value *= sqrtf(2.f/(float)(dim * (dim-1)));
	return value;
}



