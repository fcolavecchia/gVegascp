#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <unistd.h>
#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>

// includes, project
#include "hip/hip_runtime_api.h"
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "kernels.h"

int main(int argc, char* argv[])
{

   //------------------
   //  Initialization
   //------------------
   //
   // program interface:
   //   program -n "ncall0" -i "itmx0" -a "nacc" -b "nBlockSize0" -d "ndim0"
   //
   // parameters:
   //   ncall = 1024*ncall0 is the amount of function calls
   //   itmx  = itmx0 is the maximum iterations for the algorithm
   //   acc   = nacc*0.00001f is the desired accuracy
   //   nBlockSize = nBlockSize0 is the size of the CUDA block
   //   ndim = ndim0 is the dimension of the integration space

   int ncall0 = 256;
   int itmx0 = 20;
   int nacc  = 10;
   int nBlockSize0 = 256;
   int ndim0 = 6;
   int c;

   while ((c = getopt (argc, argv, "n:i:a:b:d:")) != -1)
       switch (c)
         {
         case 'n':
           ncall0 = atoi(optarg);
           break;
         case 'i':
           itmx0 = atoi(optarg);
           break;
         case 'a':
           nacc = atoi(optarg);
           break;
         case 'b':
           nBlockSize0 = atoi(optarg);
           break;
           case 'd':
             ndim0 = atoi(optarg);
             break;
         case '?':
           if (isprint (optopt))
             fprintf (stderr, "Unknown option `-%c'.\n", optopt);
           else
             fprintf (stderr,
                      "Unknown option character `\\x%x'.\n",
                      optopt);
           return 1;
         default:
           abort ();
         }

   ncall = ncall0*1024;
   itmx = itmx0;
   acc = (float)nacc*0.000001f;
   nBlockSize = nBlockSize0;
   ndim = ndim0;

   //cutilSafeCallNoSync(hipSetDevice(0));

   mds = 1;

   ng = 0;
   npg = 0;

   for (int i=0;i<ndim;i++) { //Choose the box where to integrate
      xl[i] = -1.; //lower bound
      xu[i] = 1.; //upper bound
   }

   nprn = 1;
//   nprn = -1;

   double avgi = 0.;
   double sd = 0.;
   double chi2a = 0.;

   //gVegas(avgi, sd, chi2a);
   myVegas(avgi, sd, chi2a);

   //-------------------------
   //  Print out information
   //-------------------------
   std::cout.clear();
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of Thread Block Size : "<<nBlockSize<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of dimensions        : "<<ndim<<std::endl;
   std::cout<<"# No. of func calls / iter : "<<ncall<<std::endl;
   std::cout<<"# No. of max. iterations   : "<<itmx<<std::endl;
   std::cout<<"# Desired accuracy         : "<<acc<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Answer                   : "<<avgi<<" +- "<<sd<<std::endl;
   std::cout<<"# Chisquare                : "<<chi2a<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   //Print running times!
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Function call time per iteration: " <<timeVegasCall/(double)it<<std::endl;
   std::cout<<"# Values moving time per iteration: " <<timeVegasMove/(double)it<<std::endl;
   std::cout<<"# Filling (reduce) time per iteration: " <<timeVegasFill/(double)it<<std::endl;
   std::cout<<"# Refining time per iteration: " <<timeVegasRefine/(double)it<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   //hipDeviceReset();

   return 0;
}
