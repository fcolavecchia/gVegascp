#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <unistd.h>
#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>

// includes, project
#include "hip/hip_runtime_api.h"
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "kernels.h"

int main(int argc, char* argv[])
{

   //------------------
   //  Initialization
   //------------------
   //
   // program interface:
   //   program -n "ncall0" -i "itmx0" -a "nacc" -b "nBlockSize0"
   //
   // parameters:
   //   ncall = 1024*ncall0 is the amount of function calls
   //   itmx  = itmx0 is the maximum iterations for the algorithm
   //   acc   = nacc*0.00001f is the desired accuracy
   //   nBlockSize = nBlockSize0 is the size of the CUDA block
   //

   int ncall0 = 256;
   int itmx0 = 20;
   int nacc  = 10;
   int nBlockSize0 = 256;
   int c;

   while ((c = getopt (argc, argv, "n:i:a:b:")) != -1)
       switch (c)
         {
         case 'n':
           ncall0 = atoi(optarg);
           break;
         case 'i':
           itmx0 = atoi(optarg);
           break;
         case 'a':
           nacc = atoi(optarg);
           break;
         case 'b':
           nBlockSize0 = atoi(optarg);
           break;
         case '?':
           if (isprint (optopt))
             fprintf (stderr, "Unknown option `-%c'.\n", optopt);
           else
             fprintf (stderr,
                      "Unknown option character `\\x%x'.\n",
                      optopt);
           return 1;
         default:
           abort ();
         }

   ncall = ncall0*1024;
   itmx = itmx0;
   acc = (float)nacc*0.000001f;
   nBlockSize = nBlockSize0;

   //cutilSafeCallNoSync(hipSetDevice(0));

   mds = 1;
   ndim = 6; //Dimension of integration space

   ng = 0;
   npg = 0;

   for (int i=0;i<ndim;i++) { //Choose the box where to integrate
      xl[i] = -1.; //lower bound
      xu[i] = 1.; //upper bound
   }

   nprn = 1;
//   nprn = -1;

   double avgi = 0.;
   double sd = 0.;
   double chi2a = 0.;

   gVegas(avgi, sd, chi2a);

   //-------------------------
   //  Print out information
   //-------------------------
   std::cout.clear();
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of Thread Block Size : "<<nBlockSize<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of dimensions        : "<<ndim<<std::endl;
   std::cout<<"# No. of func calls / iter : "<<ncall<<std::endl;
   std::cout<<"# No. of max. iterations   : "<<itmx<<std::endl;
   std::cout<<"# Desired accuracy         : "<<acc<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Answer                   : "<<avgi<<" +- "<<sd<<std::endl;
   std::cout<<"# Chisquare                : "<<chi2a<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   //hipDeviceReset();

   return 0;
}
