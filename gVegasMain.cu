#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <unistd.h>
#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>

// includes, project
#include "hip/hip_runtime_api.h"
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "kernels.h"

int main(int argc, char** argv)
{

   //------------------
   //  Initialization
   //------------------
   //
   // program interface:
   //   program -n="ncall0" -i="itmx0" -a="nacc" -b="nBlockSize0"
   //
   // parameters:
   //   ncall = 1024*ncall0
   //   itmx  = itmx0
   //   acc   = nacc*0.00001f
   //   nBlockSize = nBlockSize0
   //

   int ncall0 = 256;
   int itmx0 = 10;
   int nacc  = 1;
   int nBlockSize0 = 256;
   int ndim0 = 6;

   int c;

   while ((c = getopt (argc, argv, "n:i:a:b:d:")) != -1)
       switch (c)
         {
         case 'n':
           ncall0 = atoi(optarg);
           break;
         case 'i':
           itmx0 = atoi(optarg);
           break;
         case 'a':
           nacc = atoi(optarg);
           break;
         case 'b':
           nBlockSize0 = atoi(optarg);
           break;
           case 'd':
             ndim0 = atoi(optarg);
             break;
         case '?':
           if (isprint (optopt))
             fprintf (stderr, "Unknown option `-%c'.\n", optopt);
           else
             fprintf (stderr,
                      "Unknown option character `\\x%x'.\n",
                      optopt);
           return 1;
         default:
           abort ();
         }

   ncall = ncall0*1024;
   itmx = itmx0;
   acc = (float)nacc*0.00001f;
   nBlockSize = nBlockSize0;
   ndim = ndim0;

   mds = 1;

   ng = 0;
   npg = 0;

   for (int i=0;i<ndim;i++) {
      xl[i] = -1.;
      xu[i] = 1.;
   }

   nprn = 1;
//   nprn = -1;

   double avgi = 0.;
   double sd = 0.;
   double chi2a = 0.;

   gVegas(avgi, sd, chi2a);

   //-------------------------
   //  Print out information
   //-------------------------
   std::cout.clear();
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of Thread Block Size : "<<nBlockSize<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of dimensions        : "<<ndim<<std::endl;
   std::cout<<"# No. of func calls / iter : "<<ncall<<std::endl;
   std::cout<<"# No. of max. iterations   : "<<itmx<<std::endl;
   std::cout<<"# Desired accuracy         : "<<acc<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Answer                   : "<<avgi<<" +- "<<sd<<std::endl;
   std::cout<<"# Chisquare                : "<<chi2a<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   hipDeviceReset();

   //Print running times!
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Function call time per iteration: " <<timeVegasCall/(double)it<<std::endl;
   std::cout<<"# Values moving time per iteration: " <<timeVegasMove/(double)it<<std::endl;
   std::cout<<"# Filling (reduce) time per iteration: " <<timeVegasFill/(double)it<<std::endl;
   std::cout<<"# Refining time per iteration: " <<timeVegasRefine/(double)it<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   return 0;
}
