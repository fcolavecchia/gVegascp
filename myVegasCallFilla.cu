#include "hip/hip_runtime.h"
#include "vegasconst.h"
#include "vegas.h"

__global__
void myVegasCallFilla(int mds)
{
   //--------------------
   // Check the thread ID
   //--------------------
   const unsigned int tIdx  = threadIdx.x;
   const unsigned int bDimx = blockDim.x;

   const unsigned int bIdx  = blockIdx.x;
   const unsigned int gDimx = gridDim.x;
   const unsigned int bIdy  = blockIdx.y;
   //   const unsigned int gDimy = gridDim.y;

   unsigned int bid  = bIdy*gDimx+bIdx;
   const unsigned int tid = bid*bDimx+tIdx;

   __shared__ double d[ndim_max][nd_max];

   int ig = tid
   d[tid] = 0.0;
   int kg[ndim_max];
   unsigned ia[ndim_max];
   double fb = 0.0;
   double f2b = 0.0;
   __syncthreads();

   if (tid<g_nCubes) {

      for (point = 0; point < g_npg; point++){
        unsigned int tidRndm = tid * g_npg + point;

        unsigned igg = tidRndm;
        for (int j=0;j<g_ndim;j++) {
           kg[j] = igg%g_ng+1;
           igg /= g_ng;
        }

        //Generate a random point in [0,1]^ndim.
        float randm[ndim_max];
        fxorshift128(tidRndm, g_ndim, randm);

        float x[ndim_max];

        float wgt = g_xjac;
        /*
        This piece of code places the random point in the domain of integration,
        g_xi will change at every iteration as a result of the refining step, so
        the weight will change as well.
        */
        for (int j=0;j<g_ndim;j++) {
          float xo,xn,rc;
          xn = (kg[j]-randm[j])*g_dxg+1.f;
          ia[j] = (int)xn-1;
          if (ia[j]<=0) {
            xo = g_xi[j][ia[j]];
            rc = (xn-(float)(ia[j]+1))*xo;
          } else {
            xo = g_xi[j][ia[j]]-g_xi[j][ia[j]-1];
            rc = g_xi[j][ia[j]-1]+(xn-(float)(ia[j]+1))*xo;
          }
          x[j] = g_xl[j]+rc*g_dx[j];
          wgt *= xo*(float)g_nd;
        }

        double f =(double)(wgt * func(x,wgt));
        fb += f;
        double f2 = f*f;
        f2b += f*f;
        if (mds > 0){
          for (int idim = 0; idim < g_ndim; idim++) {
            atomicAdd(&d[idim][ia[idim]], f2);
          }
        }
      }

      /*When mds = -1, original code uses the data of the first element of the
      cube to store f2b in d, that won't change much if I use the last element.
      If it does, maybe we can go for a decreasing loop in npg...*/
      f2b = sqrt(f2b * g_npg);
      f2b = (f2b - fb) * (f2b - fb);
      if (mds < 0){
        for (int idim = 0; idim < g_ndim; idim++)
        atomicAdd(&d[idim][ia[idim]], f2b);
      }
      __syncthreads();
      //REDUCE TIME!!!
      
    }

}
